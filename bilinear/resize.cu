#include "hip/hip_runtime.h"
﻿/*
    Define cuda bilinear function by self
*/
#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>


__global__ void linear(const uchar* srcData, const int srcH, const int srcW, uchar* tgtData, const int tgtH, const int tgtW)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = ix + iy * tgtW;
    int idx3 = idx * 3;

    float scaleY = (float)tgtH / (float)srcH;
    float scaleX = (float)tgtW / (float)srcW;

    // (ix,iy)为目标图像坐标
    // (before_x,before_y)原图坐标
    float beforeX = float(ix + 0.5) / scaleX - 0.5;
    float beforeY = float(iy + 0.5) / scaleY - 0.5;
    // 原图像坐标四个相邻点
    // 获得变换前最近的四个顶点,取整
    int topY = static_cast<int>(beforeY);
    int bottomY = topY + 1;
    int leftX = static_cast<int>(beforeX);
    int rightX = leftX + 1;
    //计算变换前坐标的小数部分
    float u = beforeX - leftX;
    float v = beforeY - topY;

    if (ix < tgtW && iy < tgtH)
    {
        // 如果计算的原始图像的像素大于真实原始图像尺寸
        if (topY >= srcH - 1 && leftX >= srcW - 1)  //右下角
        {
            for (int k = 0; k < 3; k++)
            {
                tgtData[idx3 + k] = (1. - u) * (1. - v) * srcData[(leftX + topY * srcW) * 3 + k];
            }
        }
        else if (topY >= srcH - 1)  // 最后一行
        {
            for (int k = 0; k < 3; k++)
            {
                tgtData[idx3 + k]
                = (1. - u) * (1. - v) * srcData[(leftX + topY * srcW) * 3 + k]
                + (u) * (1. - v) * srcData[(rightX + topY * srcW) * 3 + k];
            }
        }
        else if (leftX >= srcW - 1)  // 最后一列
        {
            for (int k = 0; k < 3; k++)
            {
                tgtData[idx3 + k]
                = (1. - u) * (1. - v) * srcData[(leftX + topY * srcW) * 3 + k]
                + (1. - u) * (v) * srcData[(leftX + bottomY * srcW) * 3 + k];
            }
        }
        else  // 非最后一行或最后一列情况
        {
            for (int k = 0; k < 3; k++)
            {
                tgtData[idx3 + k]
                = (1. - u) * (1. - v) * srcData[(leftX + topY * srcW) * 3 + k]
                + (u) * (1. - v) * srcData[(rightX + topY * srcW) * 3 + k]
                + (1. - u) * (v) * srcData[(leftX + bottomY * srcW) * 3 + k]
                + u * v * srcData[(rightX + bottomY * srcW) * 3 + k];
            }
        }
    }
}


void interpolate(const cv::Mat& srcImg, cv::Mat& dstImg, const int dstHeight, const int dstWidth)
{
    int srcHeight = srcImg.rows;
    int srcWidth = srcImg.cols;
    printf("Source image width is %d, height is %d\n", srcWidth, srcHeight);
    printf("Target image width is %d, height is %d\n", dstWidth, dstHeight);
    int srcElements = srcHeight * srcWidth * 3;
    int dstElements = dstHeight * dstWidth * 3;

    // target image data on device
    uchar* dstDevData;
    hipMalloc((void**)&dstDevData, sizeof(uchar) * dstElements);
    // source images data on device
    uchar* srcDevData;
    hipMalloc((void**)&srcDevData, sizeof(uchar) * srcElements);
    double gtct_time = (double)cv::getTickCount();
    hipMemcpy(srcDevData, srcImg.data, sizeof(uchar) * srcElements, hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((dstWidth + blockSize.x - 1) / blockSize.x, (dstHeight + blockSize.y - 1) / blockSize.y);
    printf("Block(%d, %d),Grid(%d, %d).\n", blockSize.x, blockSize.y, gridSize.x, gridSize.y);

    linear<<<gridSize, blockSize>>>(srcDevData, srcHeight, srcWidth, dstDevData, dstHeight, dstWidth);

    hipMemcpy(dstImg.data, dstDevData, sizeof(uchar) * dstElements, hipMemcpyDeviceToHost);
    printf("=>need time:%.2f ms\n", ((double)cv::getTickCount() - gtct_time) / ((double)cv::getTickFrequency()) * 1000);

    hipFree(srcDevData);
    hipFree(dstDevData);
}


int main(int argc, char *argv[])
{
    if (argc != 2) {
        printf("This program need 1 argument\n");
        printf("Usage: ./resize [image path]\n");
        printf("Example: ./resize lena.jpg\n");
        return 1;
    }
    // read source image
    std::string imagePath(argv[1]);
    cv::Mat img = cv::imread(imagePath, cv::IMREAD_COLOR);

    int outputHeight = 768;
    int outputWidth = 768;
    cv::Mat outputImg(outputHeight, outputWidth, CV_8UC3, cv::Scalar(0, 0, 0));

    interpolate(img, outputImg, outputHeight, outputWidth);    

    cv::imwrite("resized.jpg", outputImg);

    return 0;
}
