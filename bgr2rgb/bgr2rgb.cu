#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>


__global__ void RGB(const uchar* srcData, uchar* tgtData, const int h, const int w)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = ix + iy * w;
    int idx3 = idx * 3;
    if (ix < w && iy < h)
    {
        tgtData[idx3] = srcData[idx3 + 2];
        tgtData[idx3 + 1] = srcData[idx3 + 1];
        tgtData[idx3 + 2] = srcData[idx3];
    }
}


void toRGB(const cv::Mat& srcImg, cv::Mat& dstImg)
{
    int w = srcImg.cols;
    int h = srcImg.rows;
    printf("Image width is %d, height is %d\n", w, h);
    int wh = w * h;
    int elements = wh * 3;

    // rgb img on device
    uchar* dstDevData;
    hipMalloc((void**)&dstDevData, sizeof(uchar) * elements);
    // bgr img on device
    uchar* srcDevData;
    hipMalloc((void**)&srcDevData, sizeof(uchar) * elements);
    double gtct_time = (double)cv::getTickCount();
    hipMemcpy(srcDevData, srcImg.data, sizeof(uchar) * elements, hipMemcpyHostToDevice);

    dim3 blockSize(32, 32);
    dim3 gridSize((w + blockSize.x - 1) / blockSize.x, (h + blockSize.y - 1) / blockSize.y);
    printf("Block(%d, %d),Grid(%d, %d).\n", blockSize.x, blockSize.y, gridSize.x, gridSize.y);

    RGB<<<gridSize, blockSize>>>(srcDevData, dstDevData, h, w);

    hipMemcpy(dstImg.data, dstDevData, sizeof(uchar) * elements, hipMemcpyDeviceToHost);
    printf("=>need time:%.2f ms\n", ((double)cv::getTickCount() - gtct_time) / ((double)cv::getTickFrequency()) * 1000);

    hipFree(srcDevData);
    hipFree(dstDevData);
}

int main(int argc, char *argv[])
{
    if (argc != 2) {
        printf("This program need 1 argument\n");
        printf("Usage: ./color2gray [image path]\n");
        printf("Example: ./color2gray lena.jpg\n");
        return 1;
    }
    // read source image
    std::string imagePath(argv[1]);
    cv::Mat img = cv::imread(imagePath, cv::IMREAD_COLOR);
    // target image
    cv::Mat outputImg(img.rows, img.cols, CV_8UC3, cv::Scalar(0, 0, 0));

    toRGB(img, outputImg);

    cv::imwrite("rgb.jpg", outputImg);

    return 0;
}
